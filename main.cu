#include <stdio.h>
#include <hip/hip_runtime.h>

// Macros to enable printf of a string macro values
#define str(s...) #s
#define xstr(s) (str(s))

/**
 * Kernel which just print's a message from thread 0, just to demonstrate device code executed,.
 */
__global__ void helloWorld() {
    if (threadIdx.x + blockDim.x * blockIdx.x == 0) {
        printf("Hello from thread %d\n", threadIdx.x + blockDim.x * blockIdx.x);
    }
}

/**
 * Main method, 
 */
int main(int argc, const char * argv[]) {

#if defined(__NVCC__)
    fprintf(stdout, "Compiled with nvcc %d.%d.%d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);
#endif
#if defined(__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ >= 12  || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 5)) 
	printf("__CUDA_ARCH_LIST__ %s\n", xstr(__CUDA_ARCH_LIST__));
#endif 

    hipError_t status = hipSuccess;
    // Get properties for the 0th device, print the name and compute capability
    int deviceIdx = 0;
    int deviceCount = 0;
    status = hipGetDeviceCount(&deviceCount);
    if (status != hipSuccess) {
        fprintf(stderr, "Error reported by cudaGetDeviceCount, %d CUDA devices found.\n  %s: %s\n", deviceCount, hipGetErrorName(status), hipGetErrorString(status));
        return EXIT_FAILURE;
    }
    if (deviceIdx >= deviceCount) {
        fprintf(stderr, "Requested device %d is not valid, %d devices found.\n", deviceIdx, deviceCount);
        return EXIT_FAILURE;
    }

    // Get and print device properties
    hipDeviceProp_t deviceProps = {};
    status = hipGetDeviceProperties(&deviceProps, deviceIdx);
    if (status != hipSuccess) {
        fprintf(stderr, "Error reported by cudaGetDeviceProperties(&deviceProps, %d) %s: %s\n", deviceIdx, hipGetErrorName(status), hipGetErrorString(status));
        return EXIT_FAILURE;
    }
    fprintf(stdout, "GPU %d: sm_%d%d %s\n", deviceIdx, deviceProps.major, deviceProps.minor, deviceProps.name);

    // Set the device
    status = hipSetDevice(deviceIdx);
    if (status != hipSuccess) {
        fprintf(stderr, "Error reported by cudaSetDevice(%d) %s: %s\n", deviceIdx, hipGetErrorName(status), hipGetErrorString(status));
        return EXIT_FAILURE;
    }

    // Initialise a cuda context, reporting any errors if failure occurs
    status = hipFree(nullptr);
    if (status != hipSuccess) {
        fprintf(stderr, "Error reported by cudaFree(nullptr).\n  %s: %s\n", hipGetErrorName(status), hipGetErrorString(status));
        return EXIT_FAILURE;
    }
    // Launch the kernel with a single thread.
    helloWorld<<<1, 1>>>();
    // Check if the kernel launch reported any errors, bue as this is pre-sync the kernel may not have executed yet 
    status = hipGetLastError();
    if (status != hipSuccess) {
        fprintf(stderr, "Error reported by cudaGetLastError() after helloWorld launch.\n  %s: %s\n", hipGetErrorName(status), hipGetErrorString(status));
        return EXIT_FAILURE;
    }
    // Sync the device 
    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        fprintf(stderr, "Error reported by cudaDeviceSynchronize().\n  %s: %s\n", hipGetErrorName(status), hipGetErrorString(status));
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
